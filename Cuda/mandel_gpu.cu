/*
 SU Project -- Taniya -- Cuda
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>

/* Bounds of the Mandelbrot set */
#define X_MIN -1.78
#define X_MAX 0.78
#define Y_MIN -0.961
#define Y_MAX 0.961

__global__ void pixel_calculation(double dx, double dy, char * pixels, int nb_iter, double x_min, double y_max, int columns);

typedef struct {

  int nb_rows, nb_columns; /* Dimensions */
  char * pixels; /* Linearized matrix of pixels */

} Image;

static void error_options () {

  fprintf (stderr, "Use : ./mandel [options]\n\n");
  fprintf (stderr, "Options \t Meaning \t\t Default val.\n\n");
  fprintf (stderr, "-n \t\t Nb iter. \t\t 100\n");
  fprintf (stderr, "-b \t\t Bounds \t\t -1.78 0.78 -0.961 0.961\n");
  fprintf (stderr, "-d \t\t Dimensions \t\t 1024 768\n");
  fprintf (stderr, "-f \t\t File \t\t /tmp/mandel.ppm\n");
  exit (1);
}

static void analyzis (int argc, char * * argv, int * nb_iter, double * x_min, double * x_max, double * y_min, double * y_max, int * width, int * height, char * * path) {

  const char * opt = "b:d:n:f:" ;
  int c ;

  /* Default values */
  * nb_iter = 100;
  * x_min = X_MIN;
  * x_max = X_MAX;
  * y_min = Y_MIN;
  * y_max = Y_MAX;
  * width = 1024;
  * height = 768;
  * path = "mandel.ppm";

  /* Analysis of arguments */
  while ((c = getopt (argc, argv, opt)) != EOF) {
    
    switch (c) {
      
    case 'b':
      sscanf (optarg, "%lf", x_min);
      sscanf (argv [optind ++], "%lf", x_max);
      sscanf (argv [optind ++], "%lf", y_min);
      sscanf (argv [optind ++], "%lf", y_max);
      break ;
    case 'd': /* width */
      sscanf (optarg, "%d", width);
      sscanf (argv [optind ++], "%d", height);
      break;
    case 'n': /* Number of iterations */
      * nb_iter = atoi (optarg);
      break;
    case 'f': /* Output file */
      * path = optarg;
      break;
    default :
      error_options ();
    };
  }  
}

static void initialization (Image * im, int nb_columns, int nb_rows) {
  im -> nb_rows = nb_rows;
  im -> nb_columns = nb_columns;
  im -> pixels = (char *) malloc (sizeof (char) * nb_rows * nb_columns); /* Space memory allocation */
} 

static void save (const Image * im, const char * path) {
  /* Image saving using the ASCII format'.PPM' */
  unsigned i;
  FILE * f = fopen (path, "w");  
  fprintf (f, "P6\n%d %d\n255\n", im -> nb_columns, im -> nb_rows); 
  for (i = 0; i < im -> nb_columns * im -> nb_rows; i ++) {
    char c = im -> pixels [i];
    fprintf (f, "%c%c%c", c, c, c); /* Monochrome weight */
  }
  fclose (f);
}

static void Compute (Image * im, int nb_iter, double x_min, double x_max, double y_min, double y_max) {
  
  double dx = (x_max - x_min) / im -> nb_columns, dy = (y_max - y_min) / im -> nb_rows; /* Discretization */
    
     int rownum = im->nb_rows, colnum = im-> nb_columns;

    dim3 blocksize(16,16,1); // 16 blocks of 16 threads each
    dim3 nblocks(rownum/16, colnum/16, 1);
    
    char * im_pixels_d;
    hipMalloc(&im_pixels_d, sizeof(char)*rownum*colnum);
    hipMemcpy(im_pixels_d, im->pixels , sizeof(char) * rownum * colnum,hipMemcpyHostToDevice);
    pixel_calculation<<< nblocks, blocksize >>> (dx, dy, im_pixels_d, nb_iter, x_min, y_max, colnum);
    
    hipMemcpy(im -> pixels, im_pixels_d, sizeof(char)*rownum*colnum,hipMemcpyDeviceToHost);
    hipFree(im_pixels_d);
}

__global__ void pixel_calculation(double dx, double dy, char * pixels, int nb_iter, double x_min, double y_max, int colnum)
{
    
    int id_x = blockIdx.x *blockDim.x + threadIdx.x;
    int id_y = blockIdx.y *blockDim.y + threadIdx.y;
    
    double a = x_min + id_y * dx, b = y_max - id_x * dy, x = 0, y = 0;
     int i=0;
      while (i < nb_iter) {
	double tmp = x;
	x = x * x - y * y + a;
	y = 2 * tmp * y + b;
	if (x * x + y * y > 4) /* Divergence ! */
	  break; 
	else
	  i++;
      }
      
      pixels [id_x*colnum+id_y] = (double) i / nb_iter * 255;
    }

int main (int argc, char * * argv) {
  
  int nb_iter, width, height; /* Degree of precision, dimensions of the image */  
  double x_min, x_max, y_min, y_max; /* Bounds of representation */
  char * path; /* File destination */
  Image im;

  analyzis(argc, argv, & nb_iter, & x_min, & x_max, & y_min, & y_max, & width, & height, & path);
  initialization (& im, width, height);
  Compute (& im, nb_iter, x_min, x_max, y_min, y_max);
  save (& im, path);
  return 0 ;
}
